#include "hip/hip_runtime.h"
#include "lrelu.h"

__global__ void lReluKern(int n, float coeff, float *ptr)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n)
    {
        return;
    }
    ptr[idx] *= ptr[idx] > 0.f ? 1.f : coeff;
}
__global__ void sigmoidKern(int n, float *ptr)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n)
    {
        return;
    }
    ptr[idx] = 1.f / (1.f + expf(-1.f * ptr[idx]));
}

void LReLU::run(hipdnnHandle_t h,
                hipdnnTensorDescriptor_t const *inputDesc, float *input,
                hipdnnTensorDescriptor_t *outputDesc, float **output,
                TagUnionExtraRet *extra)
{
    *output = input;
    size_t numElem = 0;
    checkCUDNN(cudnnGetTensorSizeInBytes(*inputDesc, &numElem));
    numElem /= sizeof(float);
    int numBlocks = (numElem + blockSize - 1) / blockSize;
    lReluKern<<<numBlocks, blockSize>>>(numElem, coeff, input);
    hipDeviceSynchronize();
}

void Sigmoid::run(hipdnnHandle_t h,
                  hipdnnTensorDescriptor_t const *inputDesc, float *input,
                  hipdnnTensorDescriptor_t *outputDesc, float **output,
                  TagUnionExtraRet *extra)
{
    *output = input;
    size_t numElem = 0;
    checkCUDNN(cudnnGetTensorSizeInBytes(*inputDesc, &numElem));
    numElem /= sizeof(float);
    int numBlocks = (numElem + blockSize - 1) / blockSize;
    sigmoidKern<<<numBlocks, blockSize>>>(numElem, input);
    hipDeviceSynchronize();
}
