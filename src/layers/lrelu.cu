#include "hip/hip_runtime.h"
#include "lrelu.h"

__global__ void lReluKern(int n, float coeff, float *ptr)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n)
    {
        return;
    }
    ptr[idx] *= ptr[idx] > 0.f ? 1.f : coeff;
}
__global__ void sigmoidKern(int n, float *ptr)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n)
    {
        return;
    }
    // if (idx & ((1 << 8)))
    // {
    //     printf("%f, ", ptr[idx]);
    // }
    ptr[idx] = 1.f / (1.f + expf(-1.f * ptr[idx]));
    // if (idx & ((1 << 8)))
    // {
    //     printf("%f\n", ptr[idx]);
    // }
}

void LReLU::run(hipdnnHandle_t h,
                hipdnnTensorDescriptor_t const *inputDesc, float *input,
                hipdnnTensorDescriptor_t *outputDesc, float **output,
                TagUnionExtraRet *extra)
{
    *output = input;
    size_t numElem = 0;
    checkCUDNN(cudnnGetTensorSizeInBytes(*inputDesc, &numElem));
    numElem /= sizeof(float);
    int numBlocks = (numElem + blockSize - 1) / blockSize;
    lReluKern<<<numBlocks, blockSize>>>(numElem, coeff, input);
    hipDeviceSynchronize();
}

void Sigmoid::run(hipdnnHandle_t h,
                  hipdnnTensorDescriptor_t const *inputDesc, float *input,
                  hipdnnTensorDescriptor_t *outputDesc, float **output,
                  TagUnionExtraRet *extra)
{
    // printf("simoid run\n");
    *output = input;
    // size_t numElem = 0;
    // checkCUDNN(cudnnGetTensorSizeInBytes(*inputDesc, &numElem));
    // // printf("tensor bytes : %i\n", numElem);
    // numElem /= sizeof(float);
    // int numBlocks = (numElem + blockSize - 1) / blockSize;
    // // printf("numBlocks is %i, blocksize is %i\n", numBlocks, blockSize);
    // sigmoidKern<<<numBlocks, blockSize>>>(numElem, input);
    // hipDeviceSynchronize();
    hipdnnActivationDescriptor_t a;
    checkCUDNN(hipdnnCreateActivationDescriptor(&a));
    checkCUDNN(hipdnnSetActivationDescriptor(a, HIPDNN_ACTIVATION_SIGMOID, HIPDNN_PROPAGATE_NAN, 0.f));
    checkCUDNN(hipdnnActivationForward(h, a, &one, *inputDesc, input, &zero, *inputDesc, input));
    checkCUDNN(hipdnnDestroyActivationDescriptor(a));
}
